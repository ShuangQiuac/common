/*
 * cudpp_hash_test.cu
 *
 *  Created on: 2019-2-21
 *      Author: qiushuang
 */
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cudpp_hash.h"
#include "type.h"
#include "utility.h"

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

#define CHECK_PTR_RETURN(ptr, ...) {									\
	if (ptr == NULL) {												\
		printf (__VA_ARGS__);										\
		printf ("Error in returned value: NULL\n");					\
		exit (1);													\
	} }

#define CHECK_KERNEL_LAUNCH() {\
		if ( hipSuccess != hipGetLastError() ) {\
		    printf( "Error in lauching kernel!\n" );\
			exit(1); }}

void test_driver (void)
{
	uint N = 100000;
	uint* key = (uint*) malloc (sizeof(uint) * N);
	CHECK_PTR_RETURN (key, "malloc host keys error!\n");
	uint* value = (uint*) malloc (sizeof(uint) * N);
	CHECK_PTR_RETURN (value, "malloc host values error!\n");
	uint* input = (uint*) malloc (sizeof(uint) * N);
	CHECK_PTR_RETURN (input, "malloc input on host error!\n");
	uint* output = (uint*) malloc (sizeof(uint) * N);
	CHECK_PTR_RETURN (output, "malloc output on host error!\n");

	for (uint i=0; i<N; i++)
	{
		key[i] = i+1;
		value[i] = key[i];
		input[i] = N-i;
	}
	uint*d_key, *d_value;
	CUDA_CHECK_RETURN(hipMalloc (&d_key, sizeof(uint) * N));
	CUDA_CHECK_RETURN(hipMalloc (&d_value, sizeof(uint) * N));
	CUDA_CHECK_RETURN(hipMemcpy(d_key, key, sizeof(uint) * N, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_value, value, sizeof(uint) * N, hipMemcpyHostToDevice));
	uint*d_input, *d_output;
	CUDA_CHECK_RETURN(hipMalloc (&d_input, sizeof(uint) * N));
	CUDA_CHECK_RETURN(hipMalloc (&d_output, sizeof(uint) * N));
	CUDA_CHECK_RETURN(hipMemcpy(d_input, input, sizeof(uint) * N, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemset(d_output, 0, sizeof(uint) * N));

	CUDPPHashTableConfig hashtabConfig = {CUDPP_BASIC_HASH_TABLE, (uint)N, 1.5};
	CUDPPHandle cudppHandle;
	cudppCreate(&cudppHandle);
	printf ("cudppHandle value: %lu\n", cudppHandle);
	CUDPPHandle tableHandle;
	cudppHashTable(cudppHandle, &tableHandle, &hashtabConfig);
	printf ("cudppHashHandle value: %lu\n", tableHandle);

	evaltime_t start, end;
	float hashInsertTime = 0;
	CUDA_CHECK_RETURN (hipDeviceSynchronize());
	gettimeofday (&start, NULL);
	cudppHashInsert(tableHandle, (void*)d_key, (void*)d_value, N);
	CUDA_CHECK_RETURN (hipDeviceSynchronize());
	gettimeofday (&end, NULL);
	hashInsertTime = (float)((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec)) / 1000;
	printf ("Hash insert  time: %f\n", hashInsertTime);
	cudppHashRetrieve(tableHandle, d_input, d_output, N);

	CUDA_CHECK_RETURN (hipMemcpy(output, d_output, sizeof(uint) * N, hipMemcpyDeviceToHost));

	for (uint i=0; i<N; i++)
	{
		printf ("key: %u, value: %u\t\tinput key: %u, retrieved value: %u\n", key[i], value[i], input[i], output[i]);
	}

	free(input);
	free(output);
	free(key);
	free(value);
	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_key);
	hipFree(d_value);
}
